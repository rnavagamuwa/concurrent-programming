#include "hip/hip_runtime.h"
// Homework 1
// Color to Greyscale Conversion

//A common way to represent color images is known as RGBA - the color
//is specified by how much Red, Green, and Blue is in it.
//The 'A' stands for Alpha and is used for transparency; it will be
//ignored in this homework.

//Each channel Red, Blue, Green, and Alpha is represented by one byte.
//Since we are using one byte for each color there are 256 different
//possible values for each color.  This means we use 4 bytes per pixel.

//Greyscale images are represented by a single intensity value per pixel
//which is one byte in size.

//To convert an image from color to grayscale one simple method is to
//set the intensity to the average of the RGB channels.  But we will
//use a more sophisticated method that takes into account how the eye 
//perceives color and weights the channels unequally.

//The eye responds most strongly to green followed by red and then blue.
//The NTSC (National Television System Committee) recommends the following
//formula for color to greyscale conversion:

//I = .299f * R + .587f * G + .114f * B

//Notice the trailing f's on the numbers which indicate that they are 
//single precision floating point constants and not double precision
//constants.

//You should fill in the kernel as well as set the block and grid sizes
//so that the entire image is processed.

#include "reference_calc.cpp"
#include "utils.h"
#include <stdio.h>

__global__
void rgba_to_greyscale(const uchar4* const rgbaImage,
                       unsigned char* const greyImage,
                       int numRows, int numCols)
{
  
  int tidX = threadIdx.x + blockIdx.x*blockDim.x; //get the actual thread index along x axis
  int tidY = threadIdx.y + blockIdx.y*blockDim.y; //get the actual thread index along y axis
  
  int numRows_perT = numRows/(gridDim.y*blockDim.y); //get the number of rows per thread
  int numCols_perT = numCols/(gridDim.x*blockDim.x); //get the number of coloumns per thread
  
   for(int r = tidX;r<numRows; r += numRows_perT){
        for(int c = tidY ; c < numCols; c+=numCols_perT){
            uchar4 rgb_val = rgbaImage[r*numCols+c]; //get the correct element from input array
            char grey_val = 0.299f*rgb_val.x + 0.587f*rgb_val.y + 0.114f*rgb_val.z; //calculate gray scale value
            greyImage[r*numCols+c] = grey_val; //store the calculated value
        }
    }
}

__global__
void rgba_to_greyscale_lightness(const uchar4* const rgbaImage,
                       unsigned char* const greyImage,
                       int numRows, int numCols)
{
    int tidX = threadIdx.x + blockIdx.x*blockDim.x;
    int tidY = threadIdx.y + blockIdx.y*blockDim.y;
   
    int numRowsPart = numRows/(gridDim.y*blockDim.y);
    int numColsPart = numCols/(gridDim.x*blockDim.x);
    for(int r = tidX;r<numRows; r += numRowsPart){
        for(int c = tidY ; c < numCols; c+=numColsPart){
            uchar4 rgb_val = rgbaImage[r*numCols+c];
            int max = 0,min =0;
            // taking the Maximum and the minimum out of all the RGB values. 
            if(rgb_val.x>rgb_val.y)
                max = rgb_val.x, min = rgb_val.y;
            else
                max = rgb_val.y,min = rgb_val.x;
            if(max<rgb_val.z)
                max = rgb_val.z;
            if(min>rgb_val.z)
                min = rgb_val.z;
                
           // set the grey_val as per the requirement.
            char grey_val = (max + min)/2 ;
            greyImage[r*numCols+c] = grey_val;
        }
    }
 
}

void your_rgba_to_greyscale(const uchar4 * const h_rgbaImage, uchar4 * const d_rgbaImage,
                            unsigned char* const d_greyImage, size_t numRows, size_t numCols)
{
  const dim3 blockSize(16, 16, 1);  //Each block has 16*16 threads
  const dim3 gridSize( 2,2, 1);  //Divides the whole image into 4 blocks
  rgba_to_greyscale<<<gridSize, blockSize>>>(d_rgbaImage, d_greyImage, numRows, numCols);
  rgba_to_greyscale_lightness<<<gridSize, blockSize>>>(d_rgbaImage, d_greyImage, numRows, numCols);
  
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}

